#include "hip/hip_runtime.h"
#include "rt_cuda.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>

namespace ray_tracing_cuda
{
bool Initialize()
{
  int count;
  hipError_t cudaStatus = hipGetDeviceCount(&count);
  if (cudaStatus != hipSuccess || count == 0)
  {
    std::cout << "Error call hipGetDeviceCount." << std::endl;
    return false;
  }

  if (hipSetDevice(0) != hipSuccess)
  {
    std::cout << "Error call hipSetDevice." << std::endl;
    return false;
  }

  hipDeviceProp_t prop;
  if (hipGetDeviceProperties(&prop, 0) != hipSuccess)
  {
    std::cout << "Error call cudaSetDeviceProperties." << std::endl;
    return false;
  }

  std::cout << "CUDA device: " << prop.name << std::endl;

  return true;
}

void RayTrace(CudaSphere * spheres, uint32_t spheresCount,
              CudaMaterial * materials, uint32_t materialsCount,
              CudaLight * lightSources, uint32_t lightSourcesCount,
              uint32_t samplesInRowCount, float3 backgroundColor,
              float3 cameraPosition, float3 cameraDirection,
              float fov, float znear, float zfar,
              uint32_t width, uint32_t height, float3 *& output)
{

}
}  // namespace ray_tracing_cuda