#include "rt_cuda.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>

namespace ray_tracing_cuda
{
bool Initialize()
{
  int count;
  hipError_t cudaStatus = hipGetDeviceCount(&count);
  if (cudaStatus != hipSuccess || count == 0)
  {
    std::cout << "Error call hipGetDeviceCount." << std::endl;
    return false;
  }

  if (hipSetDevice(0) != hipSuccess)
  {
    std::cout << "Error call hipSetDevice." << std::endl;
    return false;
  }

  hipDeviceProp_t prop;
  if (hipGetDeviceProperties(&prop, 0) != hipSuccess)
  {
    std::cout << "Error call cudaSetDeviceProperties." << std::endl;
    return false;
  }

  std::cout << "CUDA device: " << prop.name << std::endl;

  return true;
}
}  // namespace ray_tracing_cuda