#include "hip/hip_runtime.h"
#include "rt_cuda.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "cutil_math.h"
#include ""

#include <cassert>
#include <iostream>
#include <vector>

namespace ray_tracing_cuda
{
uint32_t constexpr kThreadsInRow = 8;

bool Initialize()
{
  int count;
  hipError_t cudaStatus = hipGetDeviceCount(&count);
  if (cudaStatus != hipSuccess || count == 0)
  {
    std::cout << "Error call hipGetDeviceCount." << std::endl;
    return false;
  }

  if (hipSetDevice(0) != hipSuccess)
  {
    std::cout << "Error call hipSetDevice." << std::endl;
    return false;
  }

  hipDeviceProp_t prop;
  if (hipGetDeviceProperties(&prop, 0) != hipSuccess)
  {
    std::cout << "Error call cudaSetDeviceProperties." << std::endl;
    return false;
  }

  std::cout << "CUDA device: " << prop.name << std::endl;

  return true;
}

std::vector<void *> asyncFreeMemory;

template <typename T>
class GPUPtr
{
public:
  explicit GPUPtr(uint32_t size, bool asyncFree = true) : m_size(size), m_asyncFree(asyncFree)
  {
    if (hipMalloc(&m_ptr, m_size) != hipSuccess)
      m_ptr = nullptr;

    if (m_asyncFree && m_ptr != nullptr)
      asyncFreeMemory.push_back(m_ptr);
  }

  ~GPUPtr()
  {
    if (!m_asyncFree && m_ptr != nullptr)
      hipFree(m_ptr);
  }

  operator bool() { return m_ptr != nullptr; }

  T * m_ptr = nullptr;
  uint32_t m_size;
  bool m_asyncFree;
};

struct TransferredGPUPtr
{
  void * m_ptr = nullptr;
  uint32_t m_size = 0;

  TransferredGPUPtr() = default;
  TransferredGPUPtr(void * ptr, uint32_t size) : m_ptr(ptr), m_size(size) {}

  template<typename T>
  void Set(GPUPtr<T> const & ptr)
  {
    m_ptr = ptr.m_ptr;
    m_size = ptr.m_size;
  }
};
TransferredGPUPtr transferredOutputPtr;

__device__ bool hitSphere(CudaSphere * sphere, CudaRay * ray, float tmin, float tmax, CudaHit * hit)
{
  float3 const d = ray->m_origin - sphere->m_center;
  float const a = dot(ray->m_direction, ray->m_direction);
  float const b = 2.0f * dot(d, ray->m_direction);
  float const c = dot(d, d) - sphere->m_radius * sphere->m_radius;
  float const discriminant = b * b - 4 * a * c;
  if (discriminant < 0.0f)
    return false;

  auto const sqrtD = sqrt(discriminant);
  auto const t = min((-b - sqrtD) / (2.0f * a), (-b + sqrtD) / (2.0f * a));
  if (t < tmin || t > tmax)
    return false;

  hit->m_parameterT = t;
  hit->m_position = ray->m_origin + ray->m_direction * t;
  hit->m_normal = normalize(hit->m_position - sphere->m_center);
  hit->m_materialIndex = sphere->m_materialIndex;
  return true;
}

__device__ void TraceRayGPU(CudaRay * ray, CudaSphere * spheres, uint32_t spheresCount,
                            CudaMaterial * materials, uint32_t materialsCount,
                            CudaLight * lightSources, uint32_t lightSourcesCount,
                            float3 backgroundColor, float znear, float zfar, float3 * output)
{
  CudaHit hit;
  hit.m_parameterT = zfar + 1.0f;
  bool hitFound = false;
  for (uint32_t i = 0; i < spheresCount; ++i)
  {
    CudaHit h;
    if (hitSphere(&spheres[i], ray, znear, zfar, &h))
    {
      hitFound = true;
      if (h.m_parameterT < hit.m_parameterT)
        hit = h;
    }
  }

  if (hitFound)
    *output = materials[hit.m_materialIndex].m_albedo;
  else
    *output = backgroundColor;
}

__global__ void TraceAllRaysGPU(CudaSphere * spheres, uint32_t spheresCount,
                                CudaMaterial * materials, uint32_t materialsCount,
                                CudaLight * lightSources, uint32_t lightSourcesCount,
                                float3 backgroundColor, float3 origin, float3 forward, float3 up,
                                float3 right, float2 halfScreenSize, float2 cellSize,
                                uint32_t samplesInRowCount, float invSampleCount, float znear,
                                float zfar, float3 * output)
{
  __shared__ float3 samples[kThreadsInRow][kThreadsInRow];
  int x = blockIdx.x;
  int y = blockIdx.y;

  samples[threadIdx.x][threadIdx.y] = make_float3(0.0f, 0.0f, 0.0f);
  int tx = threadIdx.x;
  while (tx < samplesInRowCount)
  {
    int ty = threadIdx.y;
    while (ty < samplesInRowCount)
    {
      float const dx = (2.0f * x / gridDim.x - 1.0f) * halfScreenSize.x;
      float const sdx = dx + cellSize.x * tx / samplesInRowCount;
      float const dy = (-2.0f * y / gridDim.y + 1.0f) * halfScreenSize.y;
      float const sdy = dy - cellSize.y * ty / samplesInRowCount;

      CudaRay ray;
      ray.m_origin = origin;
      ray.m_direction = normalize(forward * znear + up * sdy + right * sdx);

      float3 outputColor;
      TraceRayGPU(&ray, spheres, spheresCount, materials, materialsCount, lightSources,
                  lightSourcesCount, backgroundColor, znear, zfar, &outputColor);
      samples[threadIdx.x][threadIdx.y] += outputColor;

      ty += blockDim.y;
    }
    tx += blockDim.x;
  }

  // Samples reduction.
  __syncthreads();
  int j = kThreadsInRow / 2;
  while (j != 0)
  {
    if (threadIdx.x < j && threadIdx.x + j < blockDim.x)
      samples[threadIdx.x][threadIdx.y] += samples[threadIdx.x + j][threadIdx.y];
    __syncthreads();

    if (threadIdx.y < j && threadIdx.y + j < blockDim.y)
      samples[threadIdx.x][threadIdx.y] += samples[threadIdx.x][threadIdx.y + j];
    __syncthreads();
    j /= 2;
  }

  int offset = x + y * gridDim.x;
  if (threadIdx.x == 0 && threadIdx.y == 0)
    output[offset] = samples[0][0] * invSampleCount;
}

hipEvent_t RayTrace(CudaSphere * spheres, uint32_t spheresCount, CudaMaterial * materials,
                     uint32_t materialsCount, CudaLight * lightSources, uint32_t lightSourcesCount,
                     uint32_t samplesInRowCount, float3 backgroundColor, float3 cameraPosition,
                     float3 cameraDirection, float fov, float znear, float zfar, uint32_t width,
                     uint32_t height)
{
  hipEvent_t completion;
  if (hipEventCreate(&completion) != hipSuccess)
  {
    std::cout << "Error call hipEventCreate." << std::endl;
    return nullptr;
  }

  GPUPtr<CudaSphere> spheresGPU(spheresCount * sizeof(CudaSphere));
  if (!spheresGPU)
  {
    std::cout << "Error allocate GPU memory." << std::endl;
    return completion;
  }
  if (hipMemcpyAsync(spheresGPU.m_ptr, spheres, spheresGPU.m_size, hipMemcpyHostToDevice) !=
      hipSuccess)
  {
    std::cout << "Error call hipMemcpyAsync (spheresGPU)." << std::endl;
    return completion;
  }

  GPUPtr<CudaMaterial> materialsGPU(materialsCount * sizeof(CudaMaterial));
  if (!materialsGPU)
  {
    std::cout << "Error allocate GPU memory." << std::endl;
    return completion;
  }
  if (hipMemcpyAsync(materialsGPU.m_ptr, materials, materialsGPU.m_size, hipMemcpyHostToDevice) !=
      hipSuccess)
  {
    std::cout << "Error call hipMemcpyAsync (materialsGPU)." << std::endl;
    return completion;
  }

  GPUPtr<CudaLight> lightSourcesGPU(lightSourcesCount * sizeof(CudaLight));
  if (!lightSourcesGPU)
  {
    std::cout << "Error allocate GPU memory." << std::endl;
    return completion;
  }
  if (hipMemcpyAsync(lightSourcesGPU.m_ptr, lightSources, lightSourcesGPU.m_size,
                      hipMemcpyHostToDevice) != hipSuccess)
  {
    std::cout << "Error call hipMemcpyAsync (lightSourcesGPU)." << std::endl;
    return completion;
  }

  GPUPtr<float3> outputGPU(width * height * sizeof(float3));
  if (!outputGPU)
  {
    std::cout << "Error allocate GPU memory." << std::endl;
    return completion;
  }
  transferredOutputPtr.Set(outputGPU);

  static float3 kUp = make_float3(0.0f, 1.0f, 0.0f);
  auto const aspect = static_cast<float>(height) / width;

  float3 const right = cross(kUp, cameraDirection);
  float3 const up = cross(cameraDirection, right);
  float const dw = znear / tan(0.5f * fov);
  float2 const halfScreenSize = make_float2(dw, dw * aspect);
  float2 const cellSize =
      make_float2(2.0f * halfScreenSize.x / width, 2.0f * halfScreenSize.y / height);
  float const invSampleCount = 1.0f / (samplesInRowCount * samplesInRowCount);

  dim3 grids(width, height);
  dim3 threads(samplesInRowCount, samplesInRowCount);
  TraceAllRaysGPU<<<grids, kThreadsInRow>>>(
      spheresGPU.m_ptr, spheresCount, materialsGPU.m_ptr, materialsCount, lightSourcesGPU.m_ptr,
      lightSourcesCount, backgroundColor, cameraPosition, cameraDirection, up, right,
      halfScreenSize, cellSize, samplesInRowCount, invSampleCount, znear, zfar, outputGPU.m_ptr);

  auto err = hipGetLastError();
  if (err != hipSuccess)
  {
    std::cout << "Error CUDA: " << hipGetErrorString(err) << std::endl;
    return completion;
  }

  if (hipEventRecord(completion, 0) != hipSuccess)
  {
    std::cout << "Error call hipEventRecord." << std::endl;
    return completion;
  }

  return completion;
}

bool IsInProgress(hipEvent_t completion)
{
  if (hipEventQuery(completion) != hipErrorNotReady)
  {
    auto err = hipGetLastError();
    if (err != hipSuccess)
    {
      std::cout << "Error CUDA: " << hipGetErrorString(err) << std::endl;
      return true;
    }
    return false;
  }
  return true;
}

void FinishRayTrace(float * output, hipEvent_t completion)
{
  if (hipMemcpy(output, transferredOutputPtr.m_ptr, transferredOutputPtr.m_size,
                 hipMemcpyDeviceToHost) != hipSuccess)
  {
    std::cout << "Error call hipMemcpy (output)." << std::endl;
  }
    
  if (hipDeviceSynchronize() != hipSuccess)
    std::cout << "Error call hipDeviceSynchronize." << std::endl;

  auto err = hipGetLastError();
  if (err != hipSuccess)
    std::cout << "Error CUDA: " << hipGetErrorString(err) << std::endl;

  for (size_t i = 0; i < asyncFreeMemory.size(); ++i)
    hipFree(asyncFreeMemory[i]);
  asyncFreeMemory.clear();

  if (hipEventDestroy(completion) != hipSuccess)
    std::cout << "Error call hipEventDestroy." << std::endl;
}
}  // namespace ray_tracing_cuda